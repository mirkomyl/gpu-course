
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <cblas.h>

#define CHECK_CUDA_ERROR(exp) {                     \
    hipError_t ret = (exp);                        \
    if (ret != hipSuccess) {                       \
        fprintf(stderr, "[error] %s:%d: %s (%s)\n", \
            __FILE__, __LINE__,                     \
            hipGetErrorName(ret),                  \
            hipGetErrorString(ret));               \
        exit(EXIT_FAILURE);                         \
    }                                               \
}

#define CHECK_CUBLAS_ERROR(exp) {                   \
    hipblasStatus_t ret = (exp);                     \
    if (ret != HIPBLAS_STATUS_SUCCESS) {             \
        fprintf(stderr,                             \
            "[error] %s:%d: cuBLAS error\n",        \
            __FILE__, __LINE__);                    \
        exit(EXIT_FAILURE);                         \
    }                                               \
}


int main(int argc, char const **argv)
{
    // read and validate the command line arguments

    if (argc < 2) {
        fprintf(stderr, "[error] No vector lenght was supplied.\n");
        return EXIT_FAILURE;
    }

    int n = atof(argv[1]);
    if (n < 1) {
        fprintf(stderr, "[error] The vector lenght was invalid.\n");
        return EXIT_FAILURE;
    }
    
    srand(time(NULL));

    // allocate memory

    double *x, *y, *_y;
    if ((x = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector x.\n");
        return EXIT_FAILURE;
    }
    if ((y = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector y.\n");
        return EXIT_FAILURE;
    }
    if ((_y = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector _y.\n");
        return EXIT_FAILURE;
    }

    // initialize memory

    for (int i = 0; i < n; i++) {
        x[i] = 2.0 * rand()/RAND_MAX - 1.0;
        y[i] = _y[i] = 2.0 * rand()/RAND_MAX - 1.0;
    }

    // compute y <- 2 * x + y

    double alpha = 2.0;
    cblas_daxpy(n, alpha, x, 1, y, 1);

    // validate the result

    double res = 0.0;
    for (int i = 0; i < n; i++)
        res +=
            (y[i] - (alpha * x[i] + _y[i])) * (y[i] - (alpha * x[i] + _y[i]));
    printf("Residual = %e\n", sqrt(res));

    // free the allocated memory

    free(x); free(y); free(_y);

    return EXIT_SUCCESS;
}
